#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "lodepng.h"

// Pixel invert kernel
__global__ void invertPixels(unsigned char* img){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  img[i] = ~img[i];
}

int main( int argc, char ** argv) {

  int N;
  size_t pngsize;
  unsigned char *png = NULL;
  const char * filename = "lenna512x512_inv.png";
  /* Read in the image */
  lodepng_load_file(&png, &pngsize, filename);
 
  unsigned char *image = NULL;
  unsigned int width, height;
  /* Decode it into a RGB 8-bit per channel vector */
  unsigned int error = lodepng_decode24(&image, &width, &height, png, pngsize);

  /* Check if read and decode of .png went well */
  if(error != 0){
      std::cout << "error " << error << ": " << lodepng_error_text(error) << std::endl;
  }

  // Set the number of elements
  N = height * width * 3;
  // Set the needed memory
  size_t size = N * sizeof(unsigned char);
  
  // Allocate vector space in dev memory
  unsigned char* d_Array;
  hipMalloc(&d_Array, size);

  // Initialize the events
  hipEvent_t e1, e2;
  float kernel_ms, device_ms, host_ms;

  // Start first timer
  hipEventCreate(&e1);
  hipEventCreate(&e2);

  // Copy vector data to dev memory
  hipEventRecord(e1, 0);
  hipMemcpy(d_Array, image, size, hipMemcpyHostToDevice),
  hipEventRecord(e2, 0);
  hipEventSynchronize(e2);
  hipEventElapsedTime(&host_ms, e1, e2);


  // Invoke kernel
  int threadsPerBlock = 256;
  hipEventRecord(e1, 0);
  invertPixels<<<N / threadsPerBlock, threadsPerBlock>>>(d_Array);
  hipEventRecord(e2, 0);
  hipEventSynchronize(e2);
  hipEventElapsedTime(&kernel_ms, e1, e2);


  // Copy from device to host
  hipEventRecord(e1, 0);
  hipMemcpy(image, d_Array, size, hipMemcpyDeviceToHost),
  hipEventRecord(e2, 0);
  hipEventSynchronize(e2);
  hipEventElapsedTime(&device_ms, e1, e2);


  // Free device memory and events
  hipFree(d_Array);
  hipEventDestroy(e1);
  hipEventDestroy(e2);

  // Print results
  printf("Kernel: %fms\nH->D: %fms\nD->H: %fms\nTot transfer: %fms\n",
          kernel_ms, device_ms, host_ms, device_ms + host_ms);

  /* Save the result to a new .png file */
  lodepng_encode24_file("lenna512x512_orig.png", image, width, height);

  return 0;
}

