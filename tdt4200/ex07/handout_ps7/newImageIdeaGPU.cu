#include "hip/hip_runtime.h"
#include <math.h>
#include <stdbool.h>
#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include "ppmCU.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

// Image from:
// http://7-themes.com/6971875-funny-flowers-pictures.html

__global__ void performNewIdeaIterationGPU(int width, int height, float* imageOut, float* imageIn, int size) {
    int thread_x = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_y = blockIdx.y * blockDim.y + threadIdx.y;
    if (thread_x >= width || thread_y >= height) {
        return;
    }

    float sumR = 0.;
    float sumG = 0.;
    float sumB = 0.;
    int countIncluded = 0;
    for(int x = -size; x <= size; x++) {
    
        for(int y = -size; y <= size; y++) {
            int currentX = thread_x + x;
            int currentY = thread_y + y;
            
            // Check if we are outside the bounds
            if(currentX < 0)
                continue;
            if(currentX >= width)
                continue;
            if(currentY < 0)
                continue;
            if(currentY >= height)
                continue;
            
            // Now we can begin
            int offset = (width * currentY * 3) + (currentX * 3);
            sumR += imageIn[offset];
            sumG += imageIn[offset+1];
            sumB += imageIn[offset+2];
        
            // Keep track of how many values we have included
            countIncluded++;
        }
    
    }
    
    // Now we compute the final value for all colours
    float valueR = sumR / countIncluded;
    float valueG = sumG / countIncluded;
    float valueB = sumB / countIncluded;
    
    // Update the output image
    int offset = (width * thread_y * 3) + (thread_x * 3);
    imageOut[offset] = valueR;
    imageOut[offset+1] = valueG;
    imageOut[offset+2] = valueB;
}

__global__ void performNewIdeaFinalizationGPU(int width, int height, float* imageInSmall, float* imageInLarge, unsigned char* imageOut) {
    int thread_x = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_y = blockIdx.y * blockDim.y + threadIdx.y;
    if (thread_x >= width || thread_y >= height) {
        return;
    }

    int offset = width * thread_y * 3 + thread_x * 3;

    float value = (imageInLarge[offset] - imageInSmall[offset]);
    if(value > 255.0f)
        imageOut[offset] = 255;
    else if (value < -1.0f) {
        value = 257.0f+value;
        if(value > 255.0f)
            imageOut[offset] = 255;
        else
            imageOut[offset] = floorf(value);
    } else if (value > -1.0f && value < 0.0f) {
        imageOut[offset] = 0;
    } else {
        imageOut[offset] = floorf(value);
    }
    
    value = (imageInLarge[offset+1] - imageInSmall[offset+1]);
    if(value > 255.0f)
        imageOut[offset+1] = 255;
    else if (value < -1.0f) {
        value = 257.0f+value;
        if(value > 255.0f)
            imageOut[offset+1] = 255;
        else
            imageOut[offset+1] = floorf(value);
    } else if (value > -1.0f && value < 0.0f) {
        imageOut[offset+1] = 0;
    } else {
        imageOut[offset+1] = floorf(value);
    }
    
    value = (imageInLarge[offset+2] - imageInSmall[offset+2]);
    if(value > 255.0f)
        imageOut[offset+2] = 255;
    else if (value < -1.0f) {
        value = 257.0f+value;
        if(value > 255.0f)
            imageOut[offset+2] = 255;
        else
            imageOut[offset+2] = floorf(value);
    } else if (value > -1.0f && value < 0.0f) {
        imageOut[offset+2] = 0;
    } else {
        imageOut[offset+2] = floorf(value);
    }
}

__global__ void convertImageToNewFormatGPU(int width, int height, float* imageUnchanged, unsigned char* originalData) {
    // Determine unique thread ID
    int thread_x = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_y = blockIdx.y * blockDim.y + threadIdx.y;
    // Bounds check due to all-in-one formula for minimum block amount
    if (thread_x >= width || thread_y >= height) {
        return;
    }
    // Calculate offset
    int offset = width * thread_y * 3 + thread_x * 3;

    // Transform to float
    imageUnchanged[offset]   = __uint2float_rd(originalData[offset]);
    imageUnchanged[offset+1] = __uint2float_rd(originalData[offset+1]);
    imageUnchanged[offset+2] = __uint2float_rd(originalData[offset+2]);
}

typedef struct {
     float red,green,blue;
} AccuratePixel;

typedef struct {
     int x, y;
     AccuratePixel *data;
} AccurateImage;

// Convert a PPM image to a high-precision format 
AccurateImage *convertImageToNewFormat(PPMImage *image) {
    // Make a copy
    AccurateImage *imageAccurate;
    imageAccurate = (AccurateImage *)malloc(sizeof(AccurateImage));
    imageAccurate->data = (AccuratePixel*)malloc(image->x * image->y * sizeof(AccuratePixel));
    for(int i = 0; i < image->x * image->y; i++) {
        imageAccurate->data[i].red   = (float) image->data[i].red;
        imageAccurate->data[i].green = (float) image->data[i].green;
        imageAccurate->data[i].blue  = (float) image->data[i].blue;
    }
    imageAccurate->x = image->x;
    imageAccurate->y = image->y;
    
    return imageAccurate;
}

// Convert a high-precision format to a PPM image
PPMImage *convertNewFormatToPPM(AccurateImage *image) {
    // Make a copy
    PPMImage *imagePPM;
    imagePPM = (PPMImage *)malloc(sizeof(PPMImage));
    imagePPM->data = (PPMPixel*)malloc(image->x * image->y * sizeof(PPMPixel));
    for(int i = 0; i < image->x * image->y; i++) {
        imagePPM->data[i].red   = (unsigned char) image->data[i].red;
        imagePPM->data[i].green = (unsigned char) image->data[i].green;
        imagePPM->data[i].blue  = (unsigned char) image->data[i].blue;
    }
    imagePPM->x = image->x;
    imagePPM->y = image->y;
    
    return imagePPM;
}

AccurateImage *createEmptyImage(PPMImage *image){
    AccurateImage *imageAccurate;
    imageAccurate = (AccurateImage *)malloc(sizeof(AccurateImage));
    imageAccurate->data = (AccuratePixel*)malloc(image->x * image->y * sizeof(AccuratePixel));
    imageAccurate->x = image->x;
    imageAccurate->y = image->y;
    
    return imageAccurate;
}

// free memory of an AccurateImage
void freeImage(AccurateImage *image){
    free(image->data);
    free(image);
}

void performNewIdeaIteration(AccurateImage *imageOut, AccurateImage *imageIn, int size) {
    
    // Iterate over each pixel
    for(int senterX = 0; senterX < imageIn->x; senterX++) {
    
        for(int senterY = 0; senterY < imageIn->y; senterY++) {
            
            // For each pixel we compute the magic number
            float sumR = 0;
            float sumG = 0;
            float sumB = 0;
            int countIncluded = 0;
            for(int x = -size; x <= size; x++) {
            
                for(int y = -size; y <= size; y++) {
                    int currentX = senterX + x;
                    int currentY = senterY + y;
                    
                    // Check if we are outside the bounds
                    if(currentX < 0)
                        continue;
                    if(currentX >= imageIn->x)
                        continue;
                    if(currentY < 0)
                        continue;
                    if(currentY >= imageIn->y)
                        continue;
                    
                    // Now we can begin
                    int numberOfValuesInEachRow = imageIn->x; 
                    int offsetOfThePixel = (numberOfValuesInEachRow * currentY + currentX);
                    sumR += imageIn->data[offsetOfThePixel].red;
                    sumG += imageIn->data[offsetOfThePixel].green;
                    sumB += imageIn->data[offsetOfThePixel].blue;
                
                    // Keep track of how many values we have included
                    countIncluded++;
                }
            
            }
            
            // Now we compute the final value for all colours
            float valueR = sumR / countIncluded;
            float valueG = sumG / countIncluded;
            float valueB = sumB / countIncluded;
            
            // Update the output image
            int numberOfValuesInEachRow = imageOut->x; // R, G and B
            int offsetOfThePixel = (numberOfValuesInEachRow * senterY + senterX);
            imageOut->data[offsetOfThePixel].red = valueR;
            imageOut->data[offsetOfThePixel].green = valueG;
            imageOut->data[offsetOfThePixel].blue = valueB;
        }
    }
}

// Perform the final step, and save it as a ppm in imageOut
void performNewIdeaFinalization(AccurateImage *imageInSmall, AccurateImage *imageInLarge, PPMImage *imageOut) {

    
    imageOut->x = imageInSmall->x;
    imageOut->y = imageInSmall->y;
    
    for(int i = 0; i < imageInSmall->x * imageInSmall->y; i++) {
        float value = (imageInLarge->data[i].red - imageInSmall->data[i].red);
        if(value > 255.0f)
            imageOut->data[i].red = 255;
        else if (value < -1.0f) {
            value = 257.0f+value;
            if(value > 255.0f)
                imageOut->data[i].red = 255;
            else
                imageOut->data[i].red = floorf(value);
        } else if (value > -1.0f && value < 0.0f) {
            imageOut->data[i].red = 0;
        } else {
            imageOut->data[i].red = floorf(value);
        }
        
        value = (imageInLarge->data[i].green - imageInSmall->data[i].green);
        if(value > 255.0f)
            imageOut->data[i].green = 255;
        else if (value < -1.0f) {
            value = 257.0f+value;
            if(value > 255.0f)
                imageOut->data[i].green = 255;
            else
                imageOut->data[i].green = floorf(value);
        } else if (value > -1.0f && value < 0.0f) {
            imageOut->data[i].green = 0;
        } else {
            imageOut->data[i].green = floorf(value);
        }
        
        value = (imageInLarge->data[i].blue - imageInSmall->data[i].blue);
        if(value > 255.0f)
            imageOut->data[i].blue = 255;
        else if (value < -1.0f) {
            value = 257.0f+value;
            if(value > 255.0f)
                imageOut->data[i].blue = 255;
            else
                imageOut->data[i].blue = floorf(value);
        } else if (value > -1.0f && value < 0.0f) {
            imageOut->data[i].blue = 0;
        } else {
            imageOut->data[i].blue = floorf(value);
        }
    }
}

int main(int argc, char** argv) {
    
    PPMImage *image;
    
    // Determine which input stream to use
    if(argc > 1) {
        image = readPPM("flower.ppm");
    } else {
        image = readStreamPPM(stdin);
    }

    // Allocate space for final image
    PPMImage *imageOut;
    imageOut = (PPMImage *)malloc(sizeof(PPMImage));
    imageOut->data = (PPMPixel*)malloc(image->x * image->y * sizeof(PPMPixel));

    // Set up image dimensions
    int height, width;
    width = image->x;
    height = image->y;
    imageOut->x = width;
    imageOut->y = height;

    // Determine our thread block and thread grid size
    dim3 dimGrid(
        ceilf((width + 31) / 32),
        ceilf((height + 31) / 32)
    );
    dim3 dimBlock(
        32,
        32
    );

    // Cache our needed bytecounts for accurate and PPM images
    size_t imageSize = sizeof(float) * width * height * 3;
    size_t imageSizePPM = sizeof(unsigned char) * width * height * 3;

    // Declare device memory pointers
    float* devImageUnchanged;
    float* devImageBuffer;
    float* devImageSmall;
    float* devImageBig;
    unsigned char* devImageOut;

    // Allocate memory on the device
    gpuErrchk(hipMalloc((void**)&devImageUnchanged, imageSize));
    gpuErrchk(hipMalloc((void**)&devImageBuffer, imageSize));
    gpuErrchk(hipMalloc((void**)&devImageSmall, imageSize));
    gpuErrchk(hipMalloc((void**)&devImageBig, imageSize));
    gpuErrchk(hipMalloc((void**)&devImageOut, imageSizePPM));
    gpuErrchk(hipDeviceSynchronize());

    // Copy original image data
    hipMemcpy(devImageOut, image->data, imageSizePPM, hipMemcpyHostToDevice);
    gpuErrchk(hipPeekAtLastError());

    // Transform to accurate format
    convertImageToNewFormatGPU<<<dimGrid, dimBlock>>>(width, height, devImageUnchanged, devImageOut);
    gpuErrchk(hipPeekAtLastError());

    // Process the tiny case:
    performNewIdeaIterationGPU<<<dimGrid, dimBlock>>>(width, height, devImageSmall, devImageUnchanged, 2);
    gpuErrchk(hipPeekAtLastError());
    performNewIdeaIterationGPU<<<dimGrid, dimBlock>>>(width, height, devImageBuffer, devImageSmall, 2);
    gpuErrchk(hipPeekAtLastError());
    performNewIdeaIterationGPU<<<dimGrid, dimBlock>>>(width, height, devImageSmall, devImageBuffer, 2);
    gpuErrchk(hipPeekAtLastError());
    performNewIdeaIterationGPU<<<dimGrid, dimBlock>>>(width, height, devImageBuffer, devImageSmall, 2);
    gpuErrchk(hipPeekAtLastError());
    performNewIdeaIterationGPU<<<dimGrid, dimBlock>>>(width, height, devImageSmall, devImageBuffer, 2);
    gpuErrchk(hipPeekAtLastError());

    // Process the small case:
    performNewIdeaIterationGPU<<<dimGrid, dimBlock>>>(width, height, devImageBig, devImageUnchanged, 3);
    gpuErrchk(hipPeekAtLastError());
    performNewIdeaIterationGPU<<<dimGrid, dimBlock>>>(width, height, devImageBuffer, devImageBig, 3);
    gpuErrchk(hipPeekAtLastError());
    performNewIdeaIterationGPU<<<dimGrid, dimBlock>>>(width, height, devImageBig, devImageBuffer, 3);
    gpuErrchk(hipPeekAtLastError());
    performNewIdeaIterationGPU<<<dimGrid, dimBlock>>>(width, height, devImageBuffer, devImageBig, 3);
    gpuErrchk(hipPeekAtLastError());
    performNewIdeaIterationGPU<<<dimGrid, dimBlock>>>(width, height, devImageBig, devImageBuffer, 3);
    gpuErrchk(hipPeekAtLastError());

    // Save tiny case result
    performNewIdeaFinalizationGPU<<<dimGrid, dimBlock>>>(width, height, devImageSmall, devImageBig, devImageOut);
    gpuErrchk(hipPeekAtLastError());
    
    // Put finalized data in imageOut buffer for filewrite
    hipMemcpy(imageOut->data, devImageOut, imageSizePPM, hipMemcpyDeviceToHost);
    gpuErrchk(hipPeekAtLastError());

    if(argc > 1) {
        writePPM("flower_tiny.ppm", imageOut);
    } else {
        writeStreamPPM(stdout, imageOut);
    }

    // Process the medium case
    performNewIdeaIterationGPU<<<dimGrid, dimBlock>>>(width, height, devImageSmall, devImageUnchanged, 5);
    gpuErrchk(hipPeekAtLastError());
    performNewIdeaIterationGPU<<<dimGrid, dimBlock>>>(width, height, devImageBuffer, devImageSmall, 5);
    gpuErrchk(hipPeekAtLastError());
    performNewIdeaIterationGPU<<<dimGrid, dimBlock>>>(width, height, devImageSmall, devImageBuffer, 5);
    gpuErrchk(hipPeekAtLastError());
    performNewIdeaIterationGPU<<<dimGrid, dimBlock>>>(width, height, devImageBuffer, devImageSmall, 5);
    gpuErrchk(hipPeekAtLastError());
    performNewIdeaIterationGPU<<<dimGrid, dimBlock>>>(width, height, devImageSmall, devImageBuffer, 5);
    gpuErrchk(hipPeekAtLastError());
    
    // Save small case
    performNewIdeaFinalizationGPU<<<dimGrid, dimBlock>>>(width, height, devImageBig, devImageSmall, devImageOut);
    gpuErrchk(hipPeekAtLastError());

    // Put finalized data in imageOut buffer for filewrite
    hipMemcpy(imageOut->data, devImageOut, imageSizePPM, hipMemcpyDeviceToHost);
    gpuErrchk(hipPeekAtLastError());

    if(argc > 1) {
        writePPM("flower_small.ppm", imageOut);
    } else {
        writeStreamPPM(stdout, imageOut);
    }
    
    // Process the large case
    performNewIdeaIterationGPU<<<dimGrid, dimBlock>>>(width, height, devImageBig, devImageUnchanged, 8);
    gpuErrchk(hipPeekAtLastError());
    performNewIdeaIterationGPU<<<dimGrid, dimBlock>>>(width, height, devImageBuffer, devImageBig, 8);
    gpuErrchk(hipPeekAtLastError());
    performNewIdeaIterationGPU<<<dimGrid, dimBlock>>>(width, height, devImageBig, devImageBuffer, 8);
    gpuErrchk(hipPeekAtLastError());
    performNewIdeaIterationGPU<<<dimGrid, dimBlock>>>(width, height, devImageBuffer, devImageBig, 8);
    gpuErrchk(hipPeekAtLastError());
    performNewIdeaIterationGPU<<<dimGrid, dimBlock>>>(width, height, devImageBig, devImageBuffer, 8);
    gpuErrchk(hipPeekAtLastError());

    // Save the medium case   
    performNewIdeaFinalizationGPU<<<dimGrid, dimBlock>>>(width, height, devImageSmall, devImageBig, devImageOut);
    gpuErrchk(hipPeekAtLastError());

    // Put finalized data in imageOut buffer for filewrite
    hipMemcpy(imageOut->data, devImageOut, imageSizePPM, hipMemcpyDeviceToHost);
    gpuErrchk(hipPeekAtLastError());

    if(argc > 1) {
        writePPM("flower_medium.ppm", imageOut);
    } else {
        writeStreamPPM(stdout, imageOut);
    }

    // Free GPU allocated memory structures
    hipFree(devImageUnchanged);
    hipFree(devImageBuffer);
    hipFree(devImageSmall);
    hipFree(devImageBig);
    hipFree(devImageOut);

    // Free host memory structures
    free(imageOut->data);
    free(imageOut);
    free(image->data);
    free(image);
    
    return 0;
}

