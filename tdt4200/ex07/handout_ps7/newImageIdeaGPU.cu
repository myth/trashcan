#include "hip/hip_runtime.h"
#include <math.h>
#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include "ppmCU.h"

// Image from:
// http://7-themes.com/6971875-funny-flowers-pictures.html

// TODO: You must implement this
// The handout code is much simpler than the MPI/OpenMP versions
//__global__ void performNewIdeaIterationGPU( ... ) { ... }

// TODO: You should implement this
//__global__ void performNewIdeaFinalizationGPU( ... ) { ... }

// TODO: You should implement this
//__global__ void convertImageToNewFormatGPU( ... ) { ... }

// Perhaps some extra kernels will be practical as well?
//__global__ void ...GPU( ... ) { ... }

typedef struct {
     float red,green,blue;
} AccuratePixel;

typedef struct {
     int x, y;
     AccuratePixel *data;
} AccurateImage;

// Convert a PPM image to a high-precision format 
AccurateImage *convertImageToNewFormat(PPMImage *image) {
	// Make a copy
	AccurateImage *imageAccurate;
	imageAccurate = (AccurateImage *)malloc(sizeof(AccurateImage));
	imageAccurate->data = (AccuratePixel*)malloc(image->x * image->y * sizeof(AccuratePixel));
	for(int i = 0; i < image->x * image->y; i++) {
		imageAccurate->data[i].red   = (float) image->data[i].red;
		imageAccurate->data[i].green = (float) image->data[i].green;
		imageAccurate->data[i].blue  = (float) image->data[i].blue;
	}
	imageAccurate->x = image->x;
	imageAccurate->y = image->y;
	
	return imageAccurate;
}

// Convert a high-precision format to a PPM image
PPMImage *convertNewFormatToPPM(AccurateImage *image) {
	// Make a copy
	PPMImage *imagePPM;
	imagePPM = (PPMImage *)malloc(sizeof(PPMImage));
	imagePPM->data = (PPMPixel*)malloc(image->x * image->y * sizeof(PPMPixel));
	for(int i = 0; i < image->x * image->y; i++) {
		imagePPM->data[i].red   = (unsigned char) image->data[i].red;
		imagePPM->data[i].green = (unsigned char) image->data[i].green;
		imagePPM->data[i].blue  = (unsigned char) image->data[i].blue;
	}
	imagePPM->x = image->x;
	imagePPM->y = image->y;
	
	return imagePPM;
}

AccurateImage *createEmptyImage(PPMImage *image){
	AccurateImage *imageAccurate;
	imageAccurate = (AccurateImage *)malloc(sizeof(AccurateImage));
	imageAccurate->data = (AccuratePixel*)malloc(image->x * image->y * sizeof(AccuratePixel));
	imageAccurate->x = image->x;
	imageAccurate->y = image->y;
	
	return imageAccurate;
}

// free memory of an AccurateImage
void freeImage(AccurateImage *image){
	free(image->data);
	free(image);
}

void performNewIdeaIteration(AccurateImage *imageOut, AccurateImage *imageIn, int size) {
	
	// Iterate over each pixel
	for(int senterX = 0; senterX < imageIn->x; senterX++) {
	
		for(int senterY = 0; senterY < imageIn->y; senterY++) {
			
			// For each pixel we compute the magic number
			float sumR = 0;
			float sumG = 0;
			float sumB = 0;
			int countIncluded = 0;
			for(int x = -size; x <= size; x++) {
			
				for(int y = -size; y <= size; y++) {
					int currentX = senterX + x;
					int currentY = senterY + y;
					
					// Check if we are outside the bounds
					if(currentX < 0)
						continue;
					if(currentX >= imageIn->x)
						continue;
					if(currentY < 0)
						continue;
					if(currentY >= imageIn->y)
						continue;
					
					// Now we can begin
					int numberOfValuesInEachRow = imageIn->x; 
					int offsetOfThePixel = (numberOfValuesInEachRow * currentY + currentX);
					sumR += imageIn->data[offsetOfThePixel].red;
					sumG += imageIn->data[offsetOfThePixel].green;
					sumB += imageIn->data[offsetOfThePixel].blue;
				
					// Keep track of how many values we have included
					countIncluded++;
				}
			
			}
			
			// Now we compute the final value for all colours
			float valueR = sumR / countIncluded;
			float valueG = sumG / countIncluded;
			float valueB = sumB / countIncluded;
			
			// Update the output image
			int numberOfValuesInEachRow = imageOut->x; // R, G and B
			int offsetOfThePixel = (numberOfValuesInEachRow * senterY + senterX);
			imageOut->data[offsetOfThePixel].red = valueR;
			imageOut->data[offsetOfThePixel].green = valueG;
			imageOut->data[offsetOfThePixel].blue = valueB;
		}
	}
}

// Perform the final step, and save it as a ppm in imageOut
void performNewIdeaFinalization(AccurateImage *imageInSmall, AccurateImage *imageInLarge, PPMImage *imageOut) {

	
	imageOut->x = imageInSmall->x;
	imageOut->y = imageInSmall->y;
	
	for(int i = 0; i < imageInSmall->x * imageInSmall->y; i++) {
		float value = (imageInLarge->data[i].red - imageInSmall->data[i].red);
		if(value > 255.0f)
			imageOut->data[i].red = 255;
		else if (value < -1.0f) {
			value = 257.0f+value;
			if(value > 255.0f)
				imageOut->data[i].red = 255;
			else
				imageOut->data[i].red = floorf(value);
		} else if (value > -1.0f && value < 0.0f) {
			imageOut->data[i].red = 0;
		} else {
			imageOut->data[i].red = floorf(value);
		}
		
		value = (imageInLarge->data[i].green - imageInSmall->data[i].green);
		if(value > 255.0f)
			imageOut->data[i].green = 255;
		else if (value < -1.0f) {
			value = 257.0f+value;
			if(value > 255.0f)
				imageOut->data[i].green = 255;
			else
				imageOut->data[i].green = floorf(value);
		} else if (value > -1.0f && value < 0.0f) {
			imageOut->data[i].green = 0;
		} else {
			imageOut->data[i].green = floorf(value);
		}
		
		value = (imageInLarge->data[i].blue - imageInSmall->data[i].blue);
		if(value > 255.0f)
			imageOut->data[i].blue = 255;
		else if (value < -1.0f) {
			value = 257.0f+value;
			if(value > 255.0f)
				imageOut->data[i].blue = 255;
			else
				imageOut->data[i].blue = floorf(value);
		} else if (value > -1.0f && value < 0.0f) {
			imageOut->data[i].blue = 0;
		} else {
			imageOut->data[i].blue = floorf(value);
		}
	}
}

int main(int argc, char** argv) {
	
	PPMImage *image;
        
	if(argc > 1) {
		image = readPPM("flower.ppm");
	} else {
		image = readStreamPPM(stdin);
	}

	AccurateImage *imageUnchanged = convertImageToNewFormat(image); // save the unchanged image from input image
	AccurateImage *imageBuffer = createEmptyImage(image);
	AccurateImage *imageSmall = createEmptyImage(image);
	AccurateImage *imageBig = createEmptyImage(image);
	
	PPMImage *imageOut;
	imageOut = (PPMImage *)malloc(sizeof(PPMImage));
	imageOut->data = (PPMPixel*)malloc(image->x * image->y * sizeof(PPMPixel));

	// Process the tiny case:
	performNewIdeaIteration(imageSmall, imageUnchanged, 2);
	performNewIdeaIteration(imageBuffer, imageSmall, 2);
	performNewIdeaIteration(imageSmall, imageBuffer, 2);
	performNewIdeaIteration(imageBuffer, imageSmall, 2);
	performNewIdeaIteration(imageSmall, imageBuffer, 2);
	
	// Process the small case:
	performNewIdeaIteration(imageBig, imageUnchanged,3);
	performNewIdeaIteration(imageBuffer, imageBig,3);
	performNewIdeaIteration(imageBig, imageBuffer,3);
	performNewIdeaIteration(imageBuffer, imageBig,3);
	performNewIdeaIteration(imageBig, imageBuffer,3);
	
	// save tiny case result
	performNewIdeaFinalization(imageSmall,  imageBig, imageOut);
	if(argc > 1) {
		writePPM("flower_tiny.ppm", imageOut);
	} else {
		writeStreamPPM(stdout, imageOut);
	}

	
	// Process the medium case:
	performNewIdeaIteration(imageSmall, imageUnchanged, 5);
	performNewIdeaIteration(imageBuffer, imageSmall, 5);
	performNewIdeaIteration(imageSmall, imageBuffer, 5);
	performNewIdeaIteration(imageBuffer, imageSmall, 5);
	performNewIdeaIteration(imageSmall, imageBuffer, 5);
	
	// save small case
	performNewIdeaFinalization(imageBig,  imageSmall,imageOut);
	if(argc > 1) {
		writePPM("flower_small.ppm", imageOut);
	} else {
		writeStreamPPM(stdout, imageOut);
	}

	// process the large case
	performNewIdeaIteration(imageBig, imageUnchanged, 8);
	performNewIdeaIteration(imageBuffer, imageBig, 8);
	performNewIdeaIteration(imageBig, imageBuffer, 8);
	performNewIdeaIteration(imageBuffer, imageBig, 8);
	performNewIdeaIteration(imageBig, imageBuffer, 8);

	// save the medium case
	performNewIdeaFinalization(imageSmall,  imageBig, imageOut);
	if(argc > 1) {
		writePPM("flower_medium.ppm", imageOut);
	} else {
		writeStreamPPM(stdout, imageOut);
	}
	
	// free all memory structures
	freeImage(imageUnchanged);
	freeImage(imageBuffer);
	freeImage(imageSmall);
	freeImage(imageBig);
	free(imageOut->data);
	free(imageOut);
	free(image->data);
	free(image);
	
	return 0;
}

